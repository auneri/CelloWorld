
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

const int N = 16;
const int blocksize = 16;


__global__
void hello(char *a, int *b) {
	a[threadIdx.x] += b[threadIdx.x];
}


int main(int, char *[]) {
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	printf("%s", a);

	char *da;
	int *db;
	const int csize = N * sizeof(char);
	const int isize = N * sizeof(int);

	hipMalloc((void**)&da, csize);
	hipMalloc((void**)&db, isize);
	hipMemcpy(da, a, csize, hipMemcpyHostToDevice);
	hipMemcpy(db, b, isize, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	hello<<<dimGrid, dimBlock>>>(da, db);

	hipMemcpy(a, da, csize, hipMemcpyDeviceToHost);
	hipFree(da);
	hipFree(db);

	printf("%s\n", a);

	return EXIT_SUCCESS;
}
